#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/gather.h>
#include "hipblas.h"

#define START_SIZE (1<<5)
#define FINAL_SIZE (1<<15)
#define ITERATIONS (15-5+1)
#define COUNT 10
#define DATA_COUNT (ITERATIONS)
#define THREADS_PER_BLOCK (1<<5)
#define SH_DIM (THREADS_PER_BLOCK)

using namespace std;

struct init_functor
{
	float h;
	init_functor(float _h):h(_h) {}
	__device__ float operator()(float x)
	{
		return h*x;
	}
};

__global__ void gTranspose(float* da, float* db, int N, int M)
{
        int m=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;

        db[n+m*N]=da[m+n*M];
}

__global__ void gTransposeOpt(float* matr, float* matr_t, long long N, long long M)
{
        __shared__ float buffer[SH_DIM][SH_DIM+1];
        int m=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;

        buffer[threadIdx.y][threadIdx.x]=matr[m+n*M];
        __syncthreads();

        m=threadIdx.x+blockIdx.y*blockDim.x;
        n=threadIdx.y+blockIdx.x*blockDim.y;
        matr_t[m+n*N]=buffer[threadIdx.x][threadIdx.y];
}


__global__ void gInit(float* d, int s)
{
        int j=threadIdx.x+blockIdx.x*blockDim.x;
        int i=threadIdx.y+blockIdx.y*blockDim.y;
        int J=blockDim.x*gridDim.x;

        d[j+i*J]=s*(float)((j+i*J)*1.0E-3)+(1-s)*1.0f;
}

void printMatr(float* d, long long N, long long M)
{
        for(long long i=0;i<N;i++)
        {
                for(long long j=0;j<M;j++) printf("%4.2f ",d[j+i*M]);
                printf("\n");
        }
        printf("\n");
}

void printMatr(thrust::host_vector<float> d, long long N, long long M)
{
        for(long long i=0;i<N;i++)
        {
                for(long long j=0;j<M;j++) printf("%4.2f ",d[j+i*M]);
                printf("\n");
        }
        printf("\n");
}

void printResults(float time_cuda[],float time_cuda_opt[],float time_thrust[],float time_cublas[])
{
        FILE *fp=fopen("Out2.csv","w");
        long long N=START_SIZE;
        long long M=START_SIZE;
        long long index;
        fprintf(fp,"N;M;Threads per block;CUDA C;CUDA C opt;Thrust;CuBlas;\n");
        for(long long i=0;i<ITERATIONS;i++)
        {
                index=i;
                fprintf(fp,"%lli;%lli;",N,M);
                fprintf(fp,"%i;",THREADS_PER_BLOCK);
                fprintf(fp,"%f;",time_cuda[index]/COUNT);
		fprintf(fp,"%f;",time_cuda_opt[index]/COUNT);
                fprintf(fp,"%f;",time_thrust[index]/COUNT);
		fprintf(fp,"%f;",time_cublas[index]/COUNT);
		fprintf(fp,"\n");
                N<<=1;
        }
	fclose(fp);
}

int main()
{
	float time_cublas[DATA_COUNT];
	float time_cuda_opt[DATA_COUNT];
	float time_thrust[DATA_COUNT];
        float time_cuda[DATA_COUNT];
        long long N=START_SIZE;
        long long M=START_SIZE;
        float *ha,*hb,*da,*db;
	
	init_functor I(1.0E-3);
	thrust::device_vector<long long> dmap;
        thrust::device_vector<float> dVa;
        thrust::device_vector<float> dVb;
        thrust::host_vector<float> hVb;
	long long *map;

	const float alpha=1.0;
	const float beta=0.0;	

        hipEvent_t time_start,time_stop;
        hipEventCreate(&time_start);
        hipEventCreate(&time_stop);

        for(long long i=0;i<ITERATIONS;i++)
        {
                        hipMalloc((void**)&da,N*M*sizeof(float));
                        hipMalloc((void**)&db,M*N*sizeof(float));
                        gInit<<<dim3(M/THREADS_PER_BLOCK,N/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,1);
                        hipDeviceSynchronize();
                        hipEventRecord(time_start,0);
                        for(long long k=0;k<COUNT;k++)
                        {
                                gTranspose<<<dim3(M/THREADS_PER_BLOCK,N/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,db,N,M);
                                hipDeviceSynchronize();
                        }
                        hipEventRecord(time_stop,0);
                        hipEventSynchronize(time_stop);
                        hipEventElapsedTime(&(time_cuda[i]),time_start,time_stop);
			if(i==1)
                        {
                        	ha=(float*) malloc(N*M*sizeof(float));
                        	hb=(float*) malloc(M*N*sizeof(float));
                        	hipMemcpy(ha,da,N*M*sizeof(float),hipMemcpyDeviceToHost);
                        	hipMemcpy(hb,db,M*N*sizeof(float),hipMemcpyDeviceToHost);
                        	printMatr(ha,N,M);
                        	printMatr(hb,M,N);
                        	free(ha);
                        	free(hb);
                    	}
                        hipFree(db);
                        
			hipMalloc((void**)&db,M*N*sizeof(float));
			hipDeviceSynchronize();
                        hipEventRecord(time_start,0);
                        for(long long k=0;k<COUNT;k++)
                        {
                                gTransposeOpt<<<dim3(M/THREADS_PER_BLOCK,N/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,db,N,M);
                                hipDeviceSynchronize();
                        }
                        hipEventRecord(time_stop,0);
                        hipEventSynchronize(time_stop);
                        hipEventElapsedTime(&(time_cuda_opt[i]),time_start,time_stop);
			if(i==1)
                        {
				hb=(float*) malloc(M*N*sizeof(float));
                                hipMemcpy(hb,db,M*N*sizeof(float),hipMemcpyDeviceToHost);
                                printMatr(hb,M,N);
				free(hb);
                        }
			hipFree(db);

			dVa=thrust::device_vector<float>(N*M);
                        dVb=thrust::device_vector<float>(M*N);
                        hVb=thrust::host_vector<float>(N*M);
                        thrust::sequence(thrust::device,dVa.begin(),dVa.end());
                        thrust::transform(dVa.begin(),dVa.end(),dVa.begin(),I);
			map=(long long*)malloc(M*N*sizeof(long long));
			for(long long i0=0;i0<M*N;i0++) map[i0]=(i0%N)*M+(i0/N);
			dmap=thrust::device_vector<long long>(map,map+M*N);
			hipEventRecord(time_start,0);
			for(long long k=0;k<COUNT;k++) thrust::gather(dmap.begin(),dmap.end(),dVa.begin(),dVb.begin());
                        hipEventRecord(time_stop,0);
                        hipEventSynchronize(time_stop);
                        hipEventElapsedTime(&(time_thrust[i]), time_start, time_stop);
			hVb=dVb;
                        if(i==1)
                        {
                                printMatr(hVb,M,N);
                        }
			free(map);


                        hipMalloc((void**)&db,M*N*sizeof(float));
                        hipblasHandle_t cublas_handle;
                	hipblasCreate(&cublas_handle);
                	hipEventRecord(time_start,0);
                	for(long long k=0;k<COUNT;k++)
                	{
                	        hipblasSgeam(cublas_handle,HIPBLAS_OP_T,HIPBLAS_OP_T,N,M,&alpha,da,M,&beta,da,M,db,N);
              		}
                	hipEventRecord(time_stop,0);
                	hipEventSynchronize(time_stop);
                	hipEventElapsedTime(&(time_cublas[i]), time_start, time_stop);
                        hipblasDestroy(cublas_handle);
                        if(i==1)
                        {
                                hb=(float*) malloc(N*M*sizeof(float));
                                hipblasGetMatrix(M,N,sizeof(float),db,M,hb,M);
                                printMatr(hb,M,N);
                                free(hb);
                        }
                        hipFree(da);
                        hipFree(db);
                	N<<=1;
        }
        printResults(time_cuda,time_cuda_opt,time_thrust,time_cublas);
        hipEventDestroy(time_start);
        hipEventDestroy(time_stop);
}

