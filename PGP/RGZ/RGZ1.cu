#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/gather.h>
#include "hipblas.h"

#define START_SIZE (1<<5)
#define FINAL_SIZE (1<<10)
#define ITERATIONS (10-9+1)
#define COUNT 10
#define DATA_COUNT (ITERATIONS*ITERATIONS)
#define THREADS_PER_BLOCK (1<<5)

using namespace std;

struct init_functor
{
	float h;
	init_functor(float _h):h(_h) {}
	__device__ float operator()(float x)
	{
		return h*x;
	}
};

__global__ void gMultMats(float* da, float* db, float* dc,long long N, long long M, long long K)
{
        int m=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;

        float acc=0.0f;
        for(int k=0;k<K;k++) acc+=da[k+n*K]*db[m+k*M];
        dc[m+n*M]=acc;
}

__global__ void gInit(float* d, int s)
{
        int j=threadIdx.x+blockIdx.x*blockDim.x;
        int i=threadIdx.y+blockIdx.y*blockDim.y;
        int J=blockDim.x*gridDim.x;

        d[j+i*J]=s*(float)((j+i*J)*1.0E-3)+(1-s)*1.0f;
}

void printMatr(float* d, long long N, long long M)
{
        for(long long i=0;i<N;i++)
        {
                for(long long j=0;j<M;j++) printf("%4.2f ",d[j+i*M]);
                printf("\n");
        }
        printf("\n");
}

void printMatr(thrust::host_vector<float> d, long long N, long long M)
{
        for(long long i=0;i<N;i++)
        {
                for(long long j=0;j<M;j++) printf("%4.2f ",d[j+i*M]);
                printf("\n");
        }
        printf("\n");
}

void printResults(float time_cuda[],float time_thrust[],float time_cublas[])
{
        FILE *fp=fopen("Out.csv","w");
        long long N=START_SIZE;
        long long K=START_SIZE;
        long long M=START_SIZE;
        long long index;
        fprintf(fp,"N;K;M;Threads per block;CUDA C;Thrust;CuBlas;\n");
        for(long long i=0;i<ITERATIONS;i++)
        {
                K=START_SIZE;
                for(long long j=0;j<ITERATIONS;j++)
                {
                        index=i*ITERATIONS+j;
                        fprintf(fp,"%lli;%lli;%lli;",N,K,M);
                        fprintf(fp,"%i;",THREADS_PER_BLOCK);
                        fprintf(fp,"%f;",time_cuda[index]/COUNT);
                        fprintf(fp,"%f;",time_thrust[index]/COUNT);
			fprintf(fp,"%f;",time_cublas[index]/COUNT);
			fprintf(fp,"\n");
			K<<=1;
                }
                N<<=1;
        }
	fclose(fp);
}

int main()
{
	float time_cublas[DATA_COUNT];
	float time_thrust[DATA_COUNT];
        float time_cuda[DATA_COUNT];
        long long N=START_SIZE;
        long long K=START_SIZE;
        long long M=START_SIZE;
        float *ha,*hb,*hc,*da,*db,*dc;
	
	init_functor I(1.0E-3);
	thrust::device_vector<long long> dmap;
        thrust::device_vector<float> dVa;
        thrust::device_vector<float> dVb;
	thrust::device_vector<float> dVbT;
        thrust::device_vector<float> dVc;
        thrust::host_vector<float> hVc;
	thrust::device_vector<float>::iterator iterVa;
	thrust::device_vector<float>::iterator iterVbT;
	long long *map;

	const float alpha=1.0;
	const float beta=0.0;	

        hipEvent_t time_start,time_stop;
        hipEventCreate(&time_start);
        hipEventCreate(&time_stop);

        for(long long i=0;i<ITERATIONS;i++)
        {
                K=START_SIZE;
                for(long long j=0;j<ITERATIONS;j++)
                {
                        hipMalloc((void**)&da,N*K*sizeof(float));
                        hipMalloc((void**)&db,K*M*sizeof(float));
                        hipMalloc((void**)&dc,N*M*sizeof(float));
                        gInit<<<dim3(K/THREADS_PER_BLOCK,N/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,1);
                        gInit<<<dim3(M/THREADS_PER_BLOCK,K/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(db,0);
                        hipDeviceSynchronize();
                        hipEventRecord(time_start,0);
                        for(long long k=0;k<COUNT;k++)
                        {
                                gMultMats<<<dim3(M/THREADS_PER_BLOCK,N/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,db,dc,N,M,K);
                                hipDeviceSynchronize();
                        }
                        hipEventRecord(time_stop,0);
                        hipEventSynchronize(time_stop);
                        hipEventElapsedTime(&(time_cuda[i*ITERATIONS+j]),time_start,time_stop);
			if(i==1 && j==0)
                        {
                        	ha=(float*) malloc(N*K*sizeof(float));
                        	hb=(float*) malloc(K*M*sizeof(float));
                        	hc=(float*) malloc(N*M*sizeof(float));
                        	hipMemcpy(ha,da,N*K*sizeof(float),hipMemcpyDeviceToHost);
                        	hipMemcpy(hb,db,K*M*sizeof(float),hipMemcpyDeviceToHost);
                        	hipMemcpy(hc,dc,N*M*sizeof(float),hipMemcpyDeviceToHost);
                        	printMatr(ha,N,K);
                        	printMatr(hb,K,M);
                        	printMatr(hc,N,M);
                        	free(ha);
                        	free(hb);
                        	free(hc);
                    	}
                        hipFree(dc);
                        

			dVa=thrust::device_vector<float>(N*K);
                        dVb=thrust::device_vector<float>(K*M);
			dVbT=thrust::device_vector<float>(M*K);
                        dVc=thrust::device_vector<float>(N*M);
                        hVc=thrust::host_vector<float>(N*M);
                        thrust::sequence(thrust::device,dVa.begin(),dVa.end());
                        thrust::transform(dVa.begin(),dVa.end(),dVa.begin(),I);
                        thrust::fill(dVb.begin(),dVb.end(),1.0f);
			map=(long long*)malloc(M*K*sizeof(long long));
			for(long long i0=0;i0<M*K;i0++) map[i0]=(i0%K)*M+(i0/K);
			dmap=thrust::device_vector<long long>(map,map+M*K);
			thrust::gather(dmap.begin(),dmap.end(),dVb.begin(),dVbT.begin());
                        hipEventRecord(time_start,0);
			for(long long k=0;k<COUNT;k++)
                        {
				iterVa=dVa.begin();
                                for(long long i0=0;i0<N;i0++)
                                {
					iterVbT=dVbT.begin();
                                        for(long long j0=0;j0<M;j0++)
                                        {
                                                dVc[i0*M+j0]=thrust::inner_product(iterVa,iterVa+K,iterVbT,0.0f);
						iterVbT+=K;
                                        }
					iterVa+=K;
                                }
                                hVc=dVc;
                        }
                        hipEventRecord(time_stop,0);
                        hipEventSynchronize(time_stop);
                        hipEventElapsedTime(&(time_thrust[i*ITERATIONS+j]), time_start, time_stop);
                        if(i==1 && j==0)
                        {
                                printMatr(hVc,N,M);
                        }
			free(map);

                        hipMalloc((void**)&dc,N*M*sizeof(float));
                        hipblasHandle_t cublas_handle;
                	hipblasCreate(&cublas_handle);
                	hipEventRecord(time_start,0);
                	for(long long k=0;k<COUNT;k++)
                	{
                	        hipblasSgemm(cublas_handle,HIPBLAS_OP_N,HIPBLAS_OP_N,M,N,K,&alpha,db,M,da,K,&beta,dc,M);
              		}
                	hipEventRecord(time_stop,0);
                	hipEventSynchronize(time_stop);
                	hipEventElapsedTime(&(time_cublas[i*ITERATIONS+j]), time_start, time_stop);
                        hipblasDestroy(cublas_handle);
                        if(i==1 && j==0)
                        {
                                hc=(float*) malloc(N*M*sizeof(float));
                                hipblasGetMatrix(M,N,sizeof(float),dc,M,hc,M);
                                printMatr(hc,N,M);
                                free(hc);
                        }
                        hipFree(da);
                        hipFree(db);
                        hipFree(dc);
                        K<<=1;
                }
                N<<=1;
        }
        printResults(time_cuda,time_thrust,time_cublas);
        hipEventDestroy(time_start);
        hipEventDestroy(time_stop);
}

