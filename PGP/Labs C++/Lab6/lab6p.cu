#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define COEF 24
#define VERT_SIZE 2 * COEF *COEF
#define RADIUS 120.0f
#define FGSIZE 240
#define FGSHIFT FGSIZE / 2
#define MIN(A, B) (A < B ? A : B)
#define THREADS_PER_BLOCK 32
#define BLOCKS_PER_GRID MIN(32, (VERT_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)

struct Vertex
{
    float x, y, z;
};

hipArray *df_Array = 0;
texture<float, 3, hipReadModeElementType> df_tex;
__device__ Vertex vert[VERT_SIZE];

float func(float x, float y, float z)
{
    return y * y * z * z * sqrt(1 - z * z / RADIUS / RADIUS) * sqrt(1 - y * y / RADIUS / RADIUS) / RADIUS / RADIUS /
           RADIUS;
}

void calc_f(float *arr_f)
{
    for (int x = 0; x < FGSIZE; ++x)
        for (int y = 0; y < FGSIZE; ++y)
            for (int z = 0; z < FGSIZE; ++z)
                arr_f[FGSIZE * (x * FGSIZE + y) + z] = func(x - FGSHIFT, y - FGSHIFT, z - FGSHIFT);
}

void init_vertices()
{
    Vertex *temp_vert = (Vertex *)malloc(sizeof(Vertex) * VERT_SIZE);
    int i = 0;
    for (int iphi = 0; iphi < 2 * COEF; ++iphi)
        for (int ipsi = 0; ipsi < COEF; ++ipsi, ++i)
        {
            float phi = iphi * M_PI / COEF;
            float psi = ipsi * M_PI / COEF;
            temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
            temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
            temp_vert[i].z = RADIUS * cosf(psi);
        }
    hipMemcpyToSymbol(HIP_SYMBOL(vert), temp_vert, sizeof(Vertex) * VERT_SIZE, 0, hipMemcpyHostToDevice);
    free(temp_vert);
}

__device__ int getIndex(int x, int y, int z)
{
    return FGSIZE * (x * FGSIZE + y) + z;
}

__device__ float interpolate(float terms[8], float xd, float yd, float zd)
{
	float termsG1[4];
	float termsG2[2];
    termsG1[0b00] = terms[0b000] * (1 - xd) + terms[0b100] * xd;
    termsG1[0b01] = terms[0b001] * (1 - xd) + terms[0b101] * xd;

    termsG1[0b10] = terms[0b010] * (1 - xd) + terms[0b110] * xd;
    termsG1[0b11] = terms[0b011] * (1 - xd) + terms[0b111] * xd;

    termsG2[0b0] = termsG1[0b00] * (1 - yd) + termsG1[0b10] * yd;
    termsG2[0b1] = termsG1[0b01] * (1 - yd) + termsG1[0b11] * yd;

    return termsG2[0b0] * (1 - zd) + termsG2[0b1] * zd;
}

__global__ void kernelp(float *arr, float *sum)
{
    __shared__ float cache[THREADS_PER_BLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float xf = vert[tid].x+ FGSHIFT;
    float yf = vert[tid].y+ FGSHIFT;
    float zf = vert[tid].z+ FGSHIFT;

    float p;
    int x;
    int y;
    int z;
    xf = modf(xf, &p);
    x = p;
    yf = modf(yf, &p);
    y = p;
    zf = modf(zf, &p);
    z = p;

	float terms[8];
    terms[0b000] = arr[getIndex(x, y, z)];
    terms[0b001] = arr[getIndex(x, y, z + 1)];
    terms[0b010] = arr[getIndex(x, y + 1, z)];
    terms[0b011] = arr[getIndex(x, y + 1, z + 1)];
    terms[0b100] = arr[getIndex(x + 1, y, z)];
    terms[0b101] = arr[getIndex(x + 1, y, z + 1)];
    terms[0b110] = arr[getIndex(x + 1, y + 1, z)];
    terms[0b111] = arr[getIndex(x + 1, y + 1, z + 1)];

    cache[cacheIndex] = interpolate(terms, xf, yf, zf);
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (cacheIndex < s)
            cache[cacheIndex] += cache[cacheIndex + s];
        __syncthreads();
    }
    if (cacheIndex == 0)	sum[blockIdx.x] = cache[0];
}

int main(int argc, char *argv[])
{
    float *arr = (float *)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
    float *arr_dev;
    float *sum = (float *)malloc(sizeof(float) * BLOCKS_PER_GRID);
    float *sum_dev;
    float sumsSum = 0.0f;
    hipMalloc((void **)&sum_dev, sizeof(float) * BLOCKS_PER_GRID);
    hipMalloc((void **)&arr_dev, sizeof(float) * FGSIZE * FGSIZE * FGSIZE);

	calc_f(arr);
    init_vertices();
    hipMemcpy(arr_dev, arr, sizeof(float) * FGSIZE * FGSIZE * FGSIZE, hipMemcpyHostToDevice);


    kernelp<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(arr_dev, sum_dev);

    hipDeviceSynchronize();
    hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKS_PER_GRID, hipMemcpyDeviceToHost);
    for (int i = 0; i < BLOCKS_PER_GRID; ++i)
        sumsSum += sum[i];
    printf("Sump = %f\n", sumsSum * M_PI * M_PI / COEF / COEF);
    
    hipFree(sum_dev);
    free(sum);
    hipFree(arr_dev);
    free(arr);

    return 0;
}
