#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define COEF 24
#define VERT_SIZE 2 * COEF *COEF
#define RADIUS 120.0f
#define FGSIZE 240
#define FGSHIFT FGSIZE / 2
#define MIN(A, B) (A < B ? A : B)
#define THREADS_PER_BLOCK 32
#define BLOCKS_PER_GRID MIN(32, (VERT_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)

struct Vertex
{
    float x, y, z;
};

hipArray *df_Array = 0;
texture<float, 3, hipReadModeElementType> df_tex;
__constant__ Vertex vert[VERT_SIZE];

float func(float x, float y, float z)
{
    return y * y * z * z * sqrt(1 - z * z / RADIUS / RADIUS) * sqrt(1 - y * y / RADIUS / RADIUS) / RADIUS / RADIUS /
           RADIUS;
}

void calc_f(float *arr_f)
{
    for (int x = 0; x < FGSIZE; ++x)
        for (int y = 0; y < FGSIZE; ++y)
            for (int z = 0; z < FGSIZE; ++z)
                arr_f[FGSIZE * (x * FGSIZE + y) + z] = func(x - FGSHIFT, y - FGSHIFT, z - FGSHIFT);
}

void init_vertices()
{
    Vertex *temp_vert = (Vertex *)malloc(sizeof(Vertex) * VERT_SIZE);
    int i = 0;
    for (int iphi = 0; iphi < 2 * COEF; ++iphi)
        for (int ipsi = 0; ipsi < COEF; ++ipsi, ++i)
        {
            float phi = iphi * M_PI / COEF;
            float psi = ipsi * M_PI / COEF;
            temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
            temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
            temp_vert[i].z = RADIUS * cosf(psi);
        }
    hipMemcpyToSymbol(HIP_SYMBOL(vert), temp_vert, sizeof(Vertex) * VERT_SIZE, 0, hipMemcpyHostToDevice);
    free(temp_vert);
}

void load_texture(float *df_h)
{
    const hipExtent volumeSize = make_hipExtent(FGSIZE, FGSIZE, FGSIZE);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&df_Array, &channelDesc, volumeSize);
    hipMemcpy3DParms cpyParams = {0};
    cpyParams.srcPtr =
        make_hipPitchedPtr((void *)df_h, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
    cpyParams.dstArray = df_Array;
    cpyParams.extent = volumeSize;
    cpyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&cpyParams);
    df_tex.normalized = false;
    df_tex.filterMode = hipFilterModeLinear;
    df_tex.addressMode[0] = hipAddressModeClamp;
    df_tex.addressMode[1] = hipAddressModeClamp;
    df_tex.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(df_tex, df_Array, channelDesc);
}

__global__ void kernel(float *a)
{
    __shared__ float cache[THREADS_PER_BLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float x = vert[tid].x + FGSHIFT;
    float y = vert[tid].y + FGSHIFT;
    float z = vert[tid].z + FGSHIFT;
    cache[cacheIndex] = tex3D(df_tex, z,y,x);
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (cacheIndex < s)
            cache[cacheIndex] += cache[cacheIndex + s];
        __syncthreads();
    }
    if (cacheIndex == 0) a[blockIdx.x] = cache[0];
}

void release_texture()
{
    hipUnbindTexture(df_tex);
    hipFreeArray(df_Array);
}


int main(int argc, char *argv[])
{
    float *arr = (float *)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
    float *arr_dev;
    float *sum = (float *)malloc(sizeof(float) * BLOCKS_PER_GRID);
    float *sum_dev;
    float sumsSum = 0.0f;
    hipMalloc((void **)&sum_dev, sizeof(float) * BLOCKS_PER_GRID);
    hipMalloc((void **)&arr_dev, sizeof(float) * FGSIZE * FGSIZE * FGSIZE);

	calc_f(arr);
    init_vertices();
    load_texture(arr);
    
    kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(sum_dev);

    hipDeviceSynchronize();
    hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKS_PER_GRID, hipMemcpyDeviceToHost);
    for (int i = 0; i < BLOCKS_PER_GRID; ++i)
        sumsSum += sum[i];
    printf("Sum = %f\n", sumsSum * M_PI * M_PI / COEF / COEF);

    hipFree(sum_dev);
    free(sum);
    release_texture();
    hipFree(arr_dev);
    free(arr);
    return 0;
}
