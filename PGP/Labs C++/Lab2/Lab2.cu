#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void gSum(float* a, float* b, float* c)
{
        c[threadIdx.x+blockDim.x*blockIdx.x]=a[threadIdx.x+blockDim.x*blockIdx.x]+b[threadIdx.x+blockDim.x*blockIdx.x];
}

float A=-10.0;
float B=20.0;

int main(int argc, char* argv[])
{
        float *da,*ha,*db,*hb,*dc,*hc;
        long long N;
        long long threads_per_block=32;
        long long num_of_blocks;
	long long size=atoi(argv[1]);
        for(long long i=size;i<=size;i++)
        {
        	N=1<<i;
        	num_of_blocks=N/threads_per_block;
		hipMalloc((void**) &da,N*sizeof(float));
		hipMalloc((void**) &db,N*sizeof(float));
		hipMalloc((void**) &dc,N*sizeof(float));
        	ha=(float*) calloc(N,sizeof(float));
        	for(long long j=0;j<N;j++) ha[j]=rand()*(B-A)/RAND_MAX+A;
        	hb=(float*) calloc(N,sizeof(float));
        	for(long long j=0;j<N;j++) hb[j]=rand()*(B-A)/RAND_MAX+A;
        	hc=(float*) calloc(N,sizeof(float));
        	hipMemcpy(da,ha,N*sizeof(float),hipMemcpyHostToDevice);
        	hipMemcpy(db,hb,N*sizeof(float),hipMemcpyHostToDevice);
		hipDeviceSynchronize();
        	gSum<<<dim3(num_of_blocks),dim3(threads_per_block)>>>(da,db,dc);
        	hipDeviceSynchronize();
        	hipMemcpy(hc,dc,N*sizeof(float),hipMemcpyDeviceToHost);
		printf("%f %f %f\n",ha[0],hb[0],hc[0]);
		free(ha);
		free(hb);
        	free(hc);
        	hipFree(da);
        	hipFree(db);
        	hipFree(dc);
        }
        return 0;
}
