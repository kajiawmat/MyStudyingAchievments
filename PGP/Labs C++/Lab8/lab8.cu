#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "hipblas.h"

using namespace std;

#define MAX 8.0f
#define MIN -8.0f
#define THREADS_PER_BLOCK 32
#define BLOCK_DIM 32

__global__ void gMultMats(float* da, float* db, float* dc,long long N, long long M, long long K)
{
	int n=threadIdx.x+blockIdx.x*blockDim.x;
	int m=threadIdx.y+blockIdx.y*blockDim.y;
	
	float acc=0.0f;
	for(int k=0;k<K;k++) acc+=da[k+m*K]*db[n+k*N];
	dc[n+m*N]=acc;
}

__global__ void gInit(float* d, int s)
{
	int j=threadIdx.x+blockIdx.x*blockDim.x;
	int i=threadIdx.y+blockIdx.y*blockDim.y;
	int J=blockDim.x*gridDim.x;
	
	d[j+i*J]=s*(float)((j+i*J)*1.0E-3)+(1-s)*1.0f;
}

void printMatr(float* d, long long N, long long M)
{
	for(long long i=0;i<N;i++)
	{
		for(long long j=0;j<M;j++) printf("%4.2f ",d[j+i*M]);
		printf("\n");
	}
	printf("\n");
}

int main()
{
	hipEvent_t time_start, time_stop;
	hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    float time_result;
    
    float *ha,*hb,*hc,*da,*db,*dc;
    long long N=1<<6;
    long long M=1<<5;
    long long K=1<<5;
    
    for(long long i=0;i<=5;i++)
    {
        ha=(float*) malloc(M*K*sizeof(float));
        hb=(float*) malloc(K*N*sizeof(float));
        hc=(float*) malloc(M*N*sizeof(float));
        hipMalloc((void**) &da,M*K*sizeof(float));
        hipMalloc((void**) &db,K*N*sizeof(float));
        hipMalloc((void**) &dc,M*N*sizeof(float));
        
        gInit<<<dim3(K/THREADS_PER_BLOCK,M/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,1);
        hipDeviceSynchronize();
        gInit<<<dim3(N/THREADS_PER_BLOCK,K/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(db,0);
        hipDeviceSynchronize();
        hipMemcpy(ha,da,M*K*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(hb,db,K*N*sizeof(float),hipMemcpyDeviceToHost);
        hipEventRecord(time_start,0);
        gMultMats<<<dim3(N/THREADS_PER_BLOCK,M/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,db,dc,N,M,K);
        hipEventRecord(time_stop,0);
        hipEventSynchronize(time_stop);
        hipDeviceSynchronize();
        hipMemcpy(hc,dc,M*N*sizeof(float),hipMemcpyDeviceToHost);
        hipEventElapsedTime(&time_result, time_start, time_stop);
        hipDeviceSynchronize();
        //printMatr(ha,M,K);
        //printMatr(hb,K,N);
        //printMatr(hc,M,N);
        printf("M=%lli, N=%lli, K=%lli\n",M,N,K);
        printf("Time = %f\n", time_result);
	
	hipFree(dc);
        hipMalloc((void**) &dc,M*N*sizeof(float));
        hipEventRecord(time_start,0);
        hipblasHandle_t cublas_handle;
        hipblasCreate(&cublas_handle);

        const float alpha=1.0;
        const float beta=0.0;
	hipEventRecord(time_start,0);
        //cublasSgemm(cublas_handle,CUBLAS_OP_T,CUBLAS_OP_T,M,N,K,
        //&alpha,da,M,db,N,&beta,dc,M);
	hipblasSgemm(cublas_handle,HIPBLAS_OP_N,HIPBLAS_OP_N,N,M,K,
        &alpha,db,N,da,K,&beta,dc,N);
	hipEventRecord(time_stop,0);
        hipblasDestroy(cublas_handle);
        hipEventSynchronize(time_stop);
        hipDeviceSynchronize();
        hipEventElapsedTime(&time_result, time_start, time_stop);
	hipblasGetMatrix(N,M,sizeof(float),dc,N,hc,N);
        hipDeviceSynchronize();
        //printMatr(hc,M,N);
        printf("Time cuBlas = %f\n", time_result);
	
    	K<<=1;
        hipFree(da);
        hipFree(db);
        hipFree(dc);
        free(ha);
        free(hb);
        free(hc);
    }
    
    hipEventDestroy(time_start);
    hipEventDestroy(time_stop);
}
