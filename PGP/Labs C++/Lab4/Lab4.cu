#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>

using namespace std;

float A=-10.0;
float B=20.0;

__global__ void gTranspoteGlobal(float* matr, float* matr_t, long long N, long long K)
{
	int k=threadIdx.x+blockIdx.x*blockDim.x;
	int n=threadIdx.y+blockIdx.y*blockDim.y;
	
	matr_t[n+k*N]=matr[k+n*K];
}

__global__ void gInit(float* matr, long long K)
{
	int k=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;

	matr[k+n*K]=k+n*K;
}

int main(int argc, char* argv[])
{
	if(argc<2) 
	{
		printf("Too low argument in input line!");
		return -1;
	}
	long long N=atoi(argv[1]);
	long long K=atoi(argv[2]);
	long long threads_per_block=32;
	if(N%threads_per_block!=0)
	{
		printf("Wrong Input");
		return 0;
	}
	if(K%threads_per_block!=0)
        {
                printf("Wrong Input");
                return 0;
        }
	float *dmatr,*hmatr,*dmatr_t,*hmatr_t;
	hipMalloc((void**) &dmatr,N*K*sizeof(float));
	hipMalloc((void**) &dmatr_t,N*K*sizeof(float));
	hmatr=(float*) calloc(N*K,sizeof(float));
	hmatr_t=(float*) calloc(N*K,sizeof(float));
	gInit<<<dim3(K/threads_per_block,N/threads_per_block),dim3(threads_per_block,threads_per_block)>>>(dmatr,K);
	hipDeviceSynchronize();
	hipMemcpy(hmatr,dmatr,K*N*sizeof(float),hipMemcpyDeviceToHost);
	gTranspoteGlobal<<<dim3(K/threads_per_block,N/threads_per_block),dim3(threads_per_block,threads_per_block)>>>(dmatr,dmatr_t,N,K);
	hipDeviceSynchronize();
	hipMemcpy(hmatr_t,dmatr_t,K*N*sizeof(float),hipMemcpyDeviceToHost);
	for(long long i=0;i<N;i++)
        {
                for(long long j=0;j<K;j++)
                {
                        printf("%4.0f ",hmatr[j+i*K]);
                }
                printf("\n");
        }
	for(long long i=0;i<K;i++) 
	{
		for(long long j=0;j<N;j++)
		{
			printf("%4.0f ",hmatr_t[j+i*N]);
		}
		printf("\n");
	}
	hipFree(dmatr);
	hipFree(dmatr_t);
	free(hmatr);
	free(hmatr_t);
	return 0;
}
