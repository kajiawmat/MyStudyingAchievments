#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

#define COUNT_STREAM 4
#define SIZE_VECTOR (1<<15)
#define THREADS_PER_BLOCK 1
#define STEP SIZE_VECTOR/COUNT_STREAM

__global__ void gInit(float *da)
{
	long long i=threadIdx.x + blockIdx.x*blockDim.x;
	da[i]=(float)i;
}

__global__ void gSum(float *da, float *db, float *dc)
{
	long long i=threadIdx.x + blockIdx.x*blockDim.x;
	dc[i]=da[i]+db[i];
}

void printVector(float *da, long long N)
{
	for(long long i=0;i<N;i++) printf("%6.0f ",da[i]);
	printf("\n");
}

int main()
{
	hipEvent_t time_start, time_stop;
    hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    hipStream_t *streams;
    float time_result;
    
    float *ha,*hb,*hc,*da,*db,*dc;
    long long portion=THREADS_PER_BLOCK;
    
    hipHostMalloc((void**)&ha,SIZE_VECTOR*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&hb,SIZE_VECTOR*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&hc,SIZE_VECTOR*sizeof(float), hipHostMallocDefault);
    
    streams = (hipStream_t*)calloc(COUNT_STREAM,sizeof(hipStream_t));
    for(int i = 0; i < COUNT_STREAM; i++){
        hipStreamCreate(&streams[i]);
    }
    
    for(int i=0;i<=10;i++)
    {
    	hipMalloc((void**)&da,SIZE_VECTOR*sizeof(float));
    	hipMalloc((void**)&db,SIZE_VECTOR*sizeof(float));
    	hipMalloc((void**)&dc,SIZE_VECTOR*sizeof(float));
    	gInit<<<dim3(SIZE_VECTOR/portion),dim3(portion)>>>(da);
    	gInit<<<dim3(SIZE_VECTOR/portion),dim3(portion)>>>(db);
    	hipEventRecord(time_start,0);
    	for(int i=0;i<COUNT_STREAM;i++)
    	{
    		gSum<<<dim3(SIZE_VECTOR/portion),dim3(portion),0,streams[i]>>>(da,db,dc);
    	}
    	for(int i=0;i<COUNT_STREAM;i++)
		{
        	hipStreamSynchronize(streams[i]);
    	}
    	hipEventRecord(time_stop,0);
    	hipEventSynchronize(time_stop);
    	hipEventElapsedTime(&time_result, time_start, time_stop);
    	for(int i=0;i<COUNT_STREAM;i++)
    	{
    		hipMemcpyAsync(ha+(i*STEP),da+(i*STEP),sizeof(float)*STEP,hipMemcpyDeviceToHost,streams[i]);
    		hipMemcpyAsync(hb+(i*STEP),db+(i*STEP),sizeof(float)*STEP,hipMemcpyDeviceToHost,streams[i]);
    		hipMemcpyAsync(hc+(i*STEP),dc+(i*STEP),sizeof(float)*STEP,hipMemcpyDeviceToHost,streams[i]);
    	}
    	for(int i=0;i<COUNT_STREAM;i++)
		{
                hipStreamSynchronize(streams[i]);
        }
	//printVector(ha,SIZE_VECTOR);
    	//printVector(hb,SIZE_VECTOR);
    	//printVector(hc,SIZE_VECTOR);
    	printf("Portion: %lli\n",portion);
    	printf("gSum: %f\n",time_result);
    	portion<<=1;
    	hipFree(da);
    	hipFree(db);
    	hipFree(dc);
    }
    hipHostFree(ha);
    hipHostFree(hb);
    hipHostFree(hc);
    hipEventDestroy(time_start);
    hipEventDestroy(time_stop);
}
