#include <hip/hip_runtime.h>
#include <malloc.h>
#include <stdio.h>

int main()
{
	hipEvent_t time_start, time_stop;
    hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    float time_result;
    
	long long size=1<<10;
	float *ha, *hb,*da;
	for(int i=0;i<10;i++,size<<=1)
	{
		printf("SizeofVector: %lli\n",size);
		
		ha=(float*)malloc(sizeof(float)*size);
		hb=(float*)malloc(sizeof(float)*size);
		hipMalloc((void**)&da,sizeof(float)*size);
		memset(ha,21,sizeof(float)*size);
		
		hipEventRecord(time_start,0);
        hipMemcpy(da, ha, sizeof(float)*size,hipMemcpyHostToDevice);
        hipEventRecord(time_stop,0);
        hipEventSynchronize(time_stop);
        hipEventElapsedTime(&time_result, time_start, time_stop);
        printf("CopyToDevice: %f\n",time_result);
        
        hipEventRecord(time_start,0);
        hipMemcpy(hb, da, sizeof(float)*size,hipMemcpyDeviceToHost);
        hipEventRecord(time_stop,0);
        hipEventSynchronize(time_stop);
        hipEventElapsedTime(&time_result, time_start, time_stop);
        printf("CopyToDevice: %f\n",time_result);
        
        free(ha);
        free(hb);
        hipFree(da);
        
		hipHostMalloc((void**)&ha,sizeof(float)*size, hipHostMallocDefault);
		hipHostMalloc((void**)&hb,sizeof(float)*size, hipHostMallocDefault);
		hipMalloc((void**)&da,sizeof(float)*size);
		memset(ha,21,sizeof(float)*size);
		
		hipEventRecord(time_start,0);
        hipMemcpy(da, ha, sizeof(float)*size,hipMemcpyHostToDevice);
        hipEventRecord(time_stop,0);
        hipEventSynchronize(time_stop);
        hipEventElapsedTime(&time_result, time_start, time_stop);
        printf("CopyToDevice Pinned: %f\n",time_result);
        
        hipEventRecord(time_start,0);
        hipMemcpy(hb, da, sizeof(float)*size,hipMemcpyDeviceToHost);
        hipEventRecord(time_stop,0);
        hipEventSynchronize(time_stop);
        hipEventElapsedTime(&time_result, time_start, time_stop);
        printf("CopyToDevice Pinned: %f\n",time_result);
        
        hipHostFree(ha);
        hipHostFree(hb);
        hipFree(da);
        printf("\n");
	}
	
}

