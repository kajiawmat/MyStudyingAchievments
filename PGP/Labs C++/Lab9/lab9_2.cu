#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>

#define COUNT_STREAM 4
#define SIZE_VECTOR (1<<15)
#define THREADS_PER_BLOCK 1
#define STEP SIZE_VECTOR/COUNT_STREAM

__global__ void gInit(float *da)
{
	long long i=threadIdx.x + blockIdx.x*blockDim.x;
	da[i]=(float)i;
}

__global__ void gScalar(float *da, float *db, float *dsum)
{
	long long i=threadIdx.x+blockDim.x*blockIdx.x;
	__shared__ float cache[THREADS_PER_BLOCK];
    int thread=threadIdx.x;
    cache[thread]=da[i]*db[i];
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (thread < s) cache[thread] += cache[thread+s];
        __syncthreads();
    }
    if(thread==0) dsum[blockIdx.x]+=cache[0];

}

void printVector(float *da, long long N)
{
	for(long long i=0;i<N;i++) printf("%6.0f ",da[i]);
	printf("\n");
}

int main()
{
	hipEvent_t time_start, time_stop;
    hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    hipStream_t *streams;
    float time_result;
    
    float *ha,*hb,*hc,*da,*db,*dc;
    long long portion=THREADS_PER_BLOCK;
    float scalar;
    long long blocks;
    
    hipHostMalloc((void**)&ha,SIZE_VECTOR*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&hb,SIZE_VECTOR*sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&hc,SIZE_VECTOR*sizeof(float), hipHostMallocDefault);
    
    streams = (hipStream_t*)calloc(COUNT_STREAM,sizeof(hipStream_t));
    for(int i = 0; i < COUNT_STREAM; i++){
        hipStreamCreate(&streams[i]);
    }
    
    for(int i=0;i<10;i++)
    {
	blocks=SIZE_VECTOR/portion;
    	scalar=0;
    	hipMalloc((void**)&da,SIZE_VECTOR*sizeof(float));
    	hipMalloc((void**)&db,SIZE_VECTOR*sizeof(float));
    	hipMalloc((void**)&dc,blocks*sizeof(float));
    	gInit<<<dim3(blocks),dim3(portion)>>>(da);
    	gInit<<<dim3(blocks),dim3(portion)>>>(db);
	gInit<<<dim3(blocks/portion),dim3(portion)>>>(dc);
    	hipEventRecord(time_start,0);
    	for(int i=0;i<COUNT_STREAM;i++)
    	{
    		gScalar<<<dim3(blocks),dim3(portion),0,streams[i]>>>(da,db,dc);
    	}
    	for(int i=0;i<COUNT_STREAM;i++)
		{
        	hipStreamSynchronize(streams[i]);
    	}
    	hipEventRecord(time_stop,0);
    	hipEventSynchronize(time_stop);
    	hipEventElapsedTime(&time_result, time_start, time_stop);
    	for(int i=0;i<COUNT_STREAM;i++)
    	{
    		hipMemcpyAsync(ha+(i*STEP),da+(i*STEP),sizeof(float)*STEP,hipMemcpyDeviceToHost,streams[i]);
    		hipMemcpyAsync(hb+(i*STEP),db+(i*STEP),sizeof(float)*STEP,hipMemcpyDeviceToHost,streams[i]);
    		hipMemcpyAsync(hc+(i*blocks),dc+(i*blocks),sizeof(float)*blocks,hipMemcpyDeviceToHost,streams[i]);
    	}
    	for(int i=0;i<COUNT_STREAM;i++)
		{
        	hipStreamSynchronize(streams[i]);
    	}
    	for(int i=0;i<blocks;i++)
    	{
    		scalar+=hc[i];
    	}
    	//printVector(ha,SIZE_VECTOR);
    	//printVector(hb,SIZE_VECTOR);
    	//printVector(hc,SIZE_VECTOR/portion);
    	printf("Portion: %lli\n",portion);
    	printf("gScalar: %f\n",time_result);
    	portion<<=1;
    	hipFree(da);
    	hipFree(db);
    	hipFree(dc);
    }
    hipHostFree(ha);
    hipHostFree(hb);
    hipHostFree(hc);
    hipEventDestroy(time_start);
    hipEventDestroy(time_stop);
}
