#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <thrust/gather.h>
#include <thrust/execution_policy.h>

using namespace std;

#define MAX 8.0f
#define MIN -8.0f
#define ROW_SIZE 32
#define THREADS_PER_BLOCK 32
#define NUM_OF_BLOCKS 32
#define SH_DIM 32

__host__ float rand_f() {
	return ((float)rand()*(MAX-MIN)/RAND_MAX+MIN);
}


__global__ void gTranspote(float* matr, float* matr_t, long long N, long long K)
{
        int k=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;

        matr_t[k*N+n]=matr[n*K+k];
}

int main()
{
	hipEvent_t time_start, time_stop;
	hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    float time_result;
    
    float *ha,*hb,*da,*db;
    long long N=32;
	thrust::host_vector<float> hVa;
	thrust::host_vector<float> hVb;
	thrust::device_vector<float> dVa;
	thrust::device_vector<float> dVb;
	thrust::device_vector<long long> dmap;
	long long map[ROW_SIZE*(N<<4)];
    for(long long i=0;i<5;i++)
    {
        ha=(float*) malloc(ROW_SIZE*N*sizeof(float));
        hb=(float*) malloc(ROW_SIZE*N*sizeof(float));
        hipMalloc((void**) &da,ROW_SIZE*N*sizeof(float));
        hipMalloc((void**) &db,ROW_SIZE*N*sizeof(float));
        for(long long j=0;j<N;j++)
        {
            for(long long k=0;k<ROW_SIZE;k++)
            {
           		ha[j*ROW_SIZE+k]=rand_f();
           		printf("%6.2f ",ha[j*ROW_SIZE+k]);
            }
            printf("\n");    
        }
        printf("\n");
        hipMemcpy(da,ha,ROW_SIZE*N*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(time_start,0);
        gTranspote<<<dim3(ROW_SIZE/THREADS_PER_BLOCK,N/THREADS_PER_BLOCK),dim3(THREADS_PER_BLOCK,THREADS_PER_BLOCK)>>>(da,db,N,ROW_SIZE);
        hipDeviceSynchronize();
        hipMemcpy(hb,db,ROW_SIZE*N*sizeof(float),hipMemcpyDeviceToHost);
        hipEventRecord(time_stop,0);
        hipEventSynchronize(time_stop);
        hipEventElapsedTime(&time_result, time_start, time_stop);
        hipDeviceSynchronize();
        for(long long j=0;j<ROW_SIZE;j++) 
        {
        	for(long long k=0;k<N;k++) printf("%6.2f ",hb[j*N+k]);
        	printf("\n");
        }
        printf("\n");
        printf("Matrix Size = %lliX%lli\n",ROW_SIZE,N);
        printf("Time = %f\n", time_result);


		for(long long j=0;j<N*ROW_SIZE;j++) map[j]=(j%N)*ROW_SIZE+(j/N);
		hVa=thrust::host_vector<float>(ROW_SIZE*N);
        hVb=thrust::host_vector<float>(ROW_SIZE*N);
        dVa=thrust::device_vector<float>(ROW_SIZE*N);
        dVb=thrust::device_vector<float>(ROW_SIZE*N);
        dmap=thrust::device_vector<long long>(map,map+ROW_SIZE*N);

        thrust::generate(hVa.begin(),hVa.end(),rand_f);
        dVa=hVa;
        for(long long j=0;j<N;j++)
        {
            for(long long k=0;k<ROW_SIZE;k++) printf("%6.2f ",hVa[j*ROW_SIZE+k]);
            printf("\n");    
        }
        printf("\n");
    	
    	hipEventRecord(time_start,0);
    	thrust::gather(dmap.begin(),dmap.end(),dVa.begin(),dVb.begin());
    	hipEventRecord(time_stop,0);
    	hipEventSynchronize(time_stop);
    	hipEventElapsedTime(&time_result, time_start, time_stop);
    	hVb=dVb;
        for(long long j=0;j<ROW_SIZE;j++)
        {
            for(long long k=0;k<N;k++) printf("%6.2f ",hVb[j*N+k]);
            printf("\n");    
        }
        printf("\n");
    	printf("Thrust Matrix Size = %lliX%lli\n",ROW_SIZE,N);
    	printf("Thrust Time = %f\n\n", time_result);
    	
    	N<<=1;
        hipFree(da);
        hipFree(db);
        free(ha);
        free(hb);
    }
    
    hipEventDestroy(time_start);
    hipEventDestroy(time_stop);
}
