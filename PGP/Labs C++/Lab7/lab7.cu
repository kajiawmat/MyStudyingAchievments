#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>

using namespace std;

#define MAX 8.0
#define MIN -8.0
#define THREADS_PER_BLOCK 32
#define NUM_OF_BLOCKS 32

struct range_functor
{
	int max;
	range_functor(int _max):max(_max) {}
	__host__ __device__ float operator()(float x) {
		return (float)((long long)x%max);
	}
};

__global__ void gInit(float* a)
{
	a[threadIdx.x+blockDim.x*blockIdx.x]=0.0f;
}

__global__ void gScalar(float *da, float *db, float *dsum)
{
	long long i=threadIdx.x+blockDim.x*blockIdx.x;
	__shared__ float cache[THREADS_PER_BLOCK];
	int thread=threadIdx.x;
	cache[thread]=da[i]*db[i];
	__syncthreads();
	for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (thread < s) cache[thread] += cache[thread+s];
        __syncthreads();
    }
    if(thread==0) dsum[blockIdx.x]+=cache[0];
}

int main()
{
	hipEvent_t time_start, time_stop;
	hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    float time_result;
    
    float *ha,*hb,*da,*db,*hsum,*dsum;
    float sum;

    long long N=32;

    range_functor R(32);
    thrust::device_vector<float> dVa;
    thrust::device_vector<float> dVb;
    for(long long i=0;i<5;i++)
    {
	sum=0;
	hsum=(float*) malloc(sizeof(float)*NUM_OF_BLOCKS);
    	ha=(float*) malloc(N*sizeof(float));
    	hb=(float*) malloc(N*sizeof(float));
    	hipMalloc((void**) &da,N*sizeof(float));
        hipMalloc((void**) &db,N*sizeof(float));
	hipMalloc((void**) &dsum,sizeof(float)*NUM_OF_BLOCKS);
	gInit<<<NUM_OF_BLOCKS, THREADS_PER_BLOCK>>>(dsum);
    	for(long long j=0;j<N;j++)
    	{
    		//ha[j]=rand()*(MAX-MIN)/RAND_MAX+MIN;
    		//hb[j]=rand()*(MAX-MIN)/RAND_MAX+MIN;
		ha[j]=(float)(j%32);
		hb[j]=(float)(j%32);
    		printf("%6.2f ",ha[j]);
    	}
    	printf("\n");
    	for(long long j=0;j<N;j++) printf("%6.2f ",ha[j]);
    	hipMemcpy(da,ha,N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(db,hb,N*sizeof(float),hipMemcpyHostToDevice);
    	hipEventRecord(time_start,0);
    	gScalar<<<NUM_OF_BLOCKS, THREADS_PER_BLOCK>>>(da,db,dsum);
	hipDeviceSynchronize();
	hipMemcpy(hsum,dsum,sizeof(float)*NUM_OF_BLOCKS,hipMemcpyDeviceToHost);
	for(int j=0;j<NUM_OF_BLOCKS;j++) sum+=hsum[j];
        hipEventRecord(time_stop,0);
    	hipEventSynchronize(time_stop);
    	hipEventElapsedTime(&time_result, time_start, time_stop);
    	hipDeviceSynchronize();
	printf("Vector Size = %lli\n",N);
    	printf("Scalar = %f\n", sum);
    	printf("Time = %f\n", time_result);


	dVa=thrust::device_vector<float>(N);
    	dVb=thrust::device_vector<float>(N);
    	
    	thrust::sequence(thrust::device,dVa.begin(),dVa.end());
    	thrust::sequence(thrust::device,dVb.begin(),dVb.end());
    	thrust::transform(dVa.begin(),dVa.end(),dVa.begin(),R);
    	thrust::transform(dVb.begin(),dVb.end(),dVb.begin(),R);
    	
    	hipEventRecord(time_start,0);
    	sum=thrust::inner_product(dVa.begin(),dVa.end(),dVb.begin(),0.0f);
    	hipEventRecord(time_stop,0);
    	hipEventSynchronize(time_stop);
    	hipEventElapsedTime(&time_result, time_start, time_stop);
    	hipDeviceSynchronize();
    	printf("Thrust Vector Size = %lli\n",N);
    	printf("Thrust Scalar = %f\n", sum);
    	printf("Thrust Time = %f\n", time_result);

    	N<<=1;
	hipFree(da);
	hipFree(db);
	hipFree(dsum);
	free(ha);
	free(hb);
	free(hsum);
    }
    
    hipEventDestroy(time_start);
    hipEventDestroy(time_stop);
}
