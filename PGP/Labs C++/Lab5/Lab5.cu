#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>

using namespace std;

#define SH_DIM 32

float A=-10.0;
float B=20.0;

__global__ void gTranspoteGlobal(float* matr, float* matr_t, long long N, long long K)
{
        int k=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;
        matr_t[n+k*N]=matr[k+n*K];
}

__global__ void gTranspoteSharedDummy(float* matr, float* matr_t, long long N, long long K)
{
	__shared__ float buffer[SH_DIM][SH_DIM];
        int k=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;
		
	buffer[threadIdx.y][threadIdx.x]=matr[k+n*N];
	__syncthreads();

	k=threadIdx.x+blockIdx.y*blockDim.x;
	n=threadIdx.y+blockIdx.x*blockDim.y;
        matr_t[k+n*N]=buffer[threadIdx.x][threadIdx.y];
}

__global__ void gTranspoteSharedNoDummy(float* matr, float* matr_t, long long N, long long K)
{
        __shared__ float buffer[SH_DIM][SH_DIM+1];
        int k=threadIdx.x+blockIdx.x*blockDim.x;
        int n=threadIdx.y+blockIdx.y*blockDim.y;

        buffer[threadIdx.y][threadIdx.x]=matr[k+n*N];
        __syncthreads();

	k=threadIdx.x+blockIdx.y*blockDim.x;
        n=threadIdx.y+blockIdx.x*blockDim.y;
        matr_t[k+n*N]=buffer[threadIdx.x][threadIdx.y];
}

int main(int argc, char* argv[])
{
        if(argc<2)
        {
                printf("Too low argument in input line!");
                return -1;
        }
        long long N=atoi(argv[1]);
        long long K=atoi(argv[2]);
        long long threads_per_block=32;
        if(N%threads_per_block!=0)
        {
                printf("Wrong Input");
                return 0;
        }
        if(K%threads_per_block!=0)
        {
                printf("Wrong Input");
                return 0;
        }
        float *dmatr,*hmatr,*dmatr_t,*hmatr_t;
        hipMalloc((void**) &dmatr,N*K*sizeof(float));
        hipMalloc((void**) &dmatr_t,N*K*sizeof(float));
        hmatr=(float*) calloc(N*K,sizeof(float));
        hmatr_t=(float*) calloc(N*K,sizeof(float));
        for(long long i=0;i<N;i++) for(long long j=0;j<K;j++) hmatr[j+i*K]=j+i*K;//rand()*(B-A)/RAND_MAX+A;
        hipMemcpy(dmatr,hmatr,K*N*sizeof(float),hipMemcpyHostToDevice);
        gTranspoteGlobal<<<dim3(K/threads_per_block,N/threads_per_block),dim3(threads_per_block,threads_per_block)>>>(dmatr,dmatr_t,N,K);
        hipDeviceSynchronize();
        hipMemcpy(hmatr_t,dmatr_t,K*N*sizeof(float),hipMemcpyDeviceToHost);
	
	for(long long i=0;i<K;i++)
        {
                for(long long j=0;j<N;j++)
                {
                        printf("%4.0f ",hmatr_t[j+i*N]);
                }
                printf("\n");
        }

	
	hipFree(dmatr_t);
	free(hmatr_t);
	hipMalloc((void**) &dmatr_t,N*K*sizeof(float));
	hmatr_t=(float*) calloc(N*K,sizeof(float));
	gTranspoteSharedDummy<<<dim3(K/threads_per_block,N/threads_per_block),dim3(threads_per_block,threads_per_block)>>>(dmatr,dmatr_t,N,K);
        hipDeviceSynchronize();
	hipMemcpy(hmatr_t,dmatr_t,K*N*sizeof(float),hipMemcpyDeviceToHost);
	
	hipFree(dmatr_t);
        free(hmatr_t);
        hipMalloc((void**) &dmatr_t,N*K*sizeof(float));
        hmatr_t=(float*) calloc(N*K,sizeof(float));
        gTranspoteSharedNoDummy<<<dim3(K/threads_per_block,N/threads_per_block),dim3(threads_per_block,threads_per_block)>>>(dmatr,dmatr_t,N,K);
        hipDeviceSynchronize();
        hipMemcpy(hmatr_t,dmatr_t,K*N*sizeof(float),hipMemcpyDeviceToHost);

        /*for(long long i=0;i<K;i++)
        {
                for(long long j=0;j<N;j++)
                {
                        printf("%4.0f ",hmatr_t[j+i*N]);
                }
                printf("\n");
        }*/
        hipFree(dmatr);
        hipFree(dmatr_t);
        free(hmatr);
        free(hmatr_t);
        return 0;
}

