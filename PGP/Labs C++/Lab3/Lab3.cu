#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void gSum(float* a, float* b, float* c)
{
        c[threadIdx.x+blockDim.x*blockIdx.x]=a[threadIdx.x+blockDim.x*blockIdx.x]+b[threadIdx.x+blockDim.x*blockIdx.x];
}

float A=-10.0;
float B=20.0;

int main(int argc, char* argv[])
{
        float *da,*ha,*db,*hb,*dc,*hc;
        long long N=1024;
        long long threads_per_block=1<<atoi(argv[1]);
        long long num_of_blocks;
        for(long long i=0;i<=100;i++)
        {
                num_of_blocks=N/threads_per_block;
                hipMalloc((void**) &da,N*sizeof(float));
                hipMalloc((void**) &db,N*sizeof(float));
                hipMalloc((void**) &dc,N*sizeof(float));
                ha=(float*) calloc(N,sizeof(float));
                for(long long j=0;j<N;j++) ha[j]=rand()*(B-A)/RAND_MAX+A;
                hb=(float*) calloc(N,sizeof(float));
                for(long long j=0;j<N;j++) hb[j]=rand()*(B-A)/RAND_MAX+A;
                hc=(float*) calloc(N,sizeof(float));
                hipMemcpy(da,ha,N*sizeof(float),hipMemcpyHostToDevice);
                hipMemcpy(db,hb,N*sizeof(float),hipMemcpyHostToDevice);
                hipDeviceSynchronize();
                gSum<<<dim3(num_of_blocks),dim3(threads_per_block)>>>(da,db,dc);
                hipDeviceSynchronize();
                hipMemcpy(hc,dc,N*sizeof(float),hipMemcpyDeviceToHost);
                //printf("%f %f %f\n",ha[0],hb[0],hc[0]);
                free(ha);
                free(hb);
                free(hc);
                hipFree(da);
                hipFree(db);
                hipFree(dc);
        }
        return 0;
}

